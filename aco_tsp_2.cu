
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<hiprand/hiprand_kernel.h>

using namespace std;

const int CITIES=400;	
const int ANTS=400;		
const int Q=100;
const int ALPHA=1;
const int BETA=2; 
const int RHO=0.5; 
const int MAX_ITERATIONS=30;
const int WARP_SIZE=32;

int n_cities=0;
int current_iteration=0;

struct ANT_CLASS
{	
	int cur_city;
	int next_city;
	int visited_cities[CITIES];
	int tabu[CITIES];
	float path_length;
};

struct CITY_CLASS
{
	int x;
	int y;
};

CITY_CLASS city[CITIES];
ANT_CLASS ant[ANTS];
hiprandState state[ANTS];
int best=INT_MAX;
float pheromones[CITIES][CITIES];
float distances[CITIES][CITIES];
float fitness_values[CITIES][CITIES];

__global__ void initialize_random_states(hiprandState *d_random_state, int seed, int offset)
{	
	int id=threadIdx.x+blockIdx.x*blockDim.x;
	hiprand_init(seed, id, offset, &d_random_state[id]);
}

__global__ void initialize_all_values(float *d_distances, float *d_pheromones, CITY_CLASS *d_cities, int n_cities)
{	
	int ind_x=blockIdx.x*blockDim.x+threadIdx.x;
	int ind_y=blockIdx.y*blockDim.y+threadIdx.y;
	if( (ind_y<n_cities) && (ind_x<n_cities))
	{
		d_distances[ind_x+ind_y*n_cities]=0.0f;
		d_pheromones[ind_x+ind_y*n_cities]=1.0f/n_cities;

		if(ind_x!=ind_y)
		{
			d_distances[ind_x+ind_y*n_cities]=sqrt(powf(abs(d_cities[ind_y].x-d_cities[ind_x].x), 2)+powf(abs(d_cities[ind_y].y-d_cities[ind_x].y), 2));
		}
	}
}

__device__ float generate_random_value(hiprandState* rand_state, int index)
{
    hiprandState randi=rand_state[index];
    float random_value=hiprand_uniform(&randi);
    rand_state[index]=randi;
    return random_value;
}

__global__ void initialize_solution(ANT_CLASS *d_ants, int n_cities)
{	
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id<n_cities)
	{
		int j=id;
		d_ants[id].cur_city=j;
		for(int i=0;i<n_cities;i++)
		{
			d_ants[id].visited_cities[i]=0;
		}
		d_ants[id].visited_cities[j]=1;
		d_ants[id].tabu[0]=j;
		d_ants[id].path_length=0.0;
	}
}

__global__ void tau_updates(float *d_fitness_values, float *d_distances, float *d_pheromones, int n_cities)
{
	int ind_y=blockIdx.y*blockDim.y+threadIdx.y;
	int ind_x=blockIdx.x*blockDim.x+threadIdx.x;
	if(ind_y<n_cities && ind_x<n_cities)
	{
		int id=ind_y*n_cities+ind_x;
		d_fitness_values[id]=powf(d_pheromones[id], ALPHA)*powf((1.0/d_distances[id]), BETA);
	}
}

__device__ int choose_next_city(int curr_city, int n_cities, float *d_fitness_values, ANT_CLASS *d_ants, hiprandState *d_random_state)
{	
	int i=d_ants[curr_city].cur_city;
	int j;
	double prod=0.0;

	for(j=0;j<n_cities;j++)
	{
		if(d_ants[curr_city].visited_cities[j]==0)
		{
			prod+=d_fitness_values[i*n_cities+j];
		}
	}
	
	while(1)
	{
		j++;
		if(j>=n_cities)
			j=0;
		if(d_ants[curr_city].visited_cities[j]==0)
		{
			float probability=d_fitness_values[i*n_cities+j]/prod;
			float x=generate_random_value(d_random_state, i); 
			
			if(x<probability)
			{
				break;
			}
		}
	}
	
	return j;
}

__global__ void construct_solution(ANT_CLASS *d_ants, float *d_distances, float *d_fitness_values, int n_cities, hiprandState *d_random_state)
{	
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	if(id<n_cities)
	{
		for(int s=1;s<n_cities;s++)
		{	
			int j=choose_next_city(id, n_cities, d_fitness_values, d_ants, d_random_state);	
			d_ants[id].next_city=j;
			d_ants[id].visited_cities[j]=1;
			d_ants[id].tabu[s]=j;			
			d_ants[id].path_length+=d_distances[d_ants[id].cur_city*n_cities+j];
			d_ants[id].cur_city=j;
		}
	}
}

int main(int argc, char *argv[])
{	
	ifstream in;
    in.open(argv[1]);
	in>>n_cities;
	cout<<n_cities<<endl;
	int num;
	for(int i=0;i<n_cities;i++)
	{
		in>>num;	
		in>>city[i].x;
		in>>city[i].y;
	}
	
	hipEvent_t start_kernel, stop_kernel;

	hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);

	dim3 threads_per_block_2d(WARP_SIZE, WARP_SIZE, 1);
	dim3 blocks_per_grid_2d(((n_cities-1)/WARP_SIZE)+1, ((n_cities-1)/WARP_SIZE)+1, 1);
	
	int threads_per_block=WARP_SIZE;
	int blocks_per_grid=((n_cities-1)/WARP_SIZE)+1;

	hipEventRecord(start_kernel);

	float *d_distances,*d_pheromones,*d_fitness_values;
	ANT_CLASS *d_ants;
	CITY_CLASS *d_cities;
	hiprandState  *d_random_state;
	int *d_best;
	hipMalloc((void**)&d_pheromones, sizeof(float)*n_cities*n_cities);
	hipMalloc((void**)&d_distances, sizeof(float)*n_cities*n_cities);
	hipMalloc((void**)&d_ants, sizeof(ANT_CLASS)* n_cities);
	hipMalloc((void**)&d_cities, sizeof(CITY_CLASS) * n_cities);
	hipMalloc((void**)&d_fitness_values, sizeof(float) * n_cities *n_cities);
	hipMalloc( (void**) &d_random_state, sizeof(state));
	hipMalloc((void **)&d_best, sizeof(int));

	hipMemcpy(d_cities, city, sizeof(CITY_CLASS)*n_cities, hipMemcpyHostToDevice);
    hipMemcpy(d_best, &best, sizeof(int), hipMemcpyHostToDevice);	
	int seed=rand();
	initialize_random_states<<<blocks_per_grid, threads_per_block>>>(d_random_state,seed, 0);
	initialize_all_values<<<blocks_per_grid_2d, threads_per_block_2d>>>(d_distances,d_pheromones,d_cities,n_cities);
	hipMemcpy(distances,d_distances,sizeof(float) * n_cities * n_cities,hipMemcpyDeviceToHost);
	hipMemcpy(pheromones,d_pheromones,sizeof(float) * n_cities * n_cities,hipMemcpyDeviceToHost);
	
	do
	{		
		cout<<"The best path length is: "<<best<<endl;
		hipDeviceSynchronize();
		initialize_solution<<<blocks_per_grid, threads_per_block>>>(d_ants,n_cities);
		hipDeviceSynchronize();
		tau_updates<<<blocks_per_grid_2d, threads_per_block_2d>>>(d_fitness_values, d_distances, d_pheromones, n_cities);
		hipDeviceSynchronize();
		construct_solution<<<blocks_per_grid, threads_per_block>>>(d_ants, d_distances, d_fitness_values, n_cities, d_random_state);
		hipDeviceSynchronize();
		hipMemcpy(ant, d_ants, sizeof(ANT_CLASS)*n_cities, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		current_iteration++;
		
	}while(current_iteration<MAX_ITERATIONS);

	hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);

    float runTime;
	hipEventElapsedTime(&runTime, start_kernel, stop_kernel);

	printf("RunTime: %fms\n", runTime);

	return 0;
}

