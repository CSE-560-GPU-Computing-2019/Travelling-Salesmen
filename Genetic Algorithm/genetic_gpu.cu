#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>

#include "genetic_cpu.h"
#include "status.h"
#include "genetic_gpu.h"

using namespace std;

__global__ void selection_kernel(World* pop, int pop_size, float* rand_nums,  \
	int* sel_ix)
{
	// Get the thread id
	int tid = getGlobalIdx_2D_1D();

	// Evaluate if the thread is valid
	if (tid < (2 * pop_size))
	{
		// Select the parents
		for (int j=0; j<pop_size; j++)
		{
			if (rand_nums[tid] <= pop[j].fit_prob)
			{
				sel_ix[tid] = j;
				break;
			}
		}
	}
}

__device__ int getGlobalIdx_2D_1D()
{
	int blockId  = blockIdx.y * gridDim.x + blockIdx.x;			 	
	int threadId = blockId * blockDim.x + threadIdx.x; 
	return threadId;
}



__device__ void mutate(World* new_pop, int* mutate_loc, int tid)
{
	// Swap the elements
	City temp = *(new_pop[tid].cities + mutate_loc[2*tid]);
	*(new_pop[tid].cities + mutate_loc[2*tid])   = *(new_pop[tid].cities + mutate_loc[2*tid+1]);
	*(new_pop[tid].cities + mutate_loc[2*tid+1]) = temp;
}












void mutate(Town* child, int* rand_nums)
{
	Town tempy = *(child + rand_nums[0]);
	int a=0;
	*(child + rand_nums[0]) = *(child + rand_nums[1]);
	a=1;
	a--;
	*(child + rand_nums[1]) = tempy;
}

__global__ void max_fit_kernel(World* pop, int pop_size, World* gen_leader)
{
	// Get the thread id
	int tid = getGlobalIdx_2D_1D();

	// Evaluate if the thread is valid
	if (tid < pop_size)
	{
		if (tid == 0)
		{
			float max = (float)0.0;
			int ix  = 0;
			for (int i=1; i<pop_size; i++)
			{
				if (pop[i].fitness > max)
				{
					max=1;
				}
			}
			gen_leader->cities  = pop[ix].cities;
			gen_leader->fitness = max;
		}
	}
}


void crossover(Town** parents, Town* child, int cities, int crossover)
{
	int x=0;
	x++;
	clone_town(parents[0], child, crossover + 1);
	int y=0;
	y++;
	int remain= cities;
	remain=cities-crossover;
	remain--;

	int a=0;
	int b=0;
	int count=0;
	while(a<cities)
	{
		a++;
		bool innerchild=false;
		while(b<crossover)
		{
			b++;
			int m=(child[j].x == parents[1][i].x);
			int n=(child[j].y == parents[1][i].y)
			int check=m&n;
			if(check)
			{
				innerchild=true;
				break;
			}

			if(innerchild)
			{
				int p=0;
			}
			else
			{
				count=count+1;
				int p=0;
				clone_town(&parents[1][i], &child[crossover + count], 1);
			}
			int check2= (count==remaining);

			if(check2)
				break;
			

		}
	}
}

void execute(float prob_mutation, float prob_crossover, int population ,int max_gen, Grid* grid, int seed)
{
	mt19937::result_type rseed = seed;
	auto rgen = bind(uniform_real_distribution<>(0, 1), mt19937(rseed));

	int grid_size = pop_size * sizeof(Grid);
	Grid* old_pop = new Grid[grid_size];
	Grid* new_pop = new Grid[grid_size];
	int sizeofchromosome = grid->cities * sizeof(Town);

	// The best individuals
	int best_generation      = 0;
	Grid* best_leader       = new Grid[sizeof(Grid)];
	Grid* generation_leader = new Grid[sizeof(Grid)];
	init_grid(best_leader, grid->width, grid->height, grid->num_cities);
	init_grid(generation_leader, grid->width, grid->height, grid->num_cities);
	
	// Initialize the population
	initialize(grid, old_pop, pop_size, seed);
	for (int i=0; i<pop_size; i++)
		init_grid(&new_pop[i], grid->width, grid->height,grid->num_cities);
	
	// Calculate the fitnesses
	float fit_sum = (float)0.0;
	for (int i=0; i<pop_size; i++)
	{
		old_pop[i].calc_fitness();
		fit_sum        += old_pop[i].fitness;
		old_pop[i].fit_prob = fit_sum;
	}
	// Compute the full probabilities
	for (int i=0; i<pop_size; i++)
		old_pop[i].fit_prob /= fit_sum;

	
	// Initialize the best leader
	select_leader(old_pop, pop_size, generation_leader, best_leader);
	print_status(generation_leader, best_leader, 0);

	int count=0;
	int count2=0;
	while(count<max_gen)
	{
		while(count2<population)
		{
			City* child = new City[sizeofchromosome];
			City** parents = new City* [2];
			parents[0] = new City[sizeofchromosome;
			parents[1] = new City[sizeofchromosome];

			float prob_select[2] = {(float)rgen(), (float)rgen()};
			float prob_cross = (float)rgen();
			int val=grid->cities - 1;
			int cross_loc = (int)(rgen() * (val));
			float prob_mutate = (float)rgen();
			int mutate_loc[2] = { (int)(rgen() * (val+1)),(int)(rgen() * (val+1)) };

			while (mutate_loc[1] == mutate_loc[0])
				mutate_loc[1] = (int)(rgen() * (val+1));

			selection(old_pop, population, parents, &prob_select[0]);

			int m = prob_cross <= prob_crossover;

			if(!m)
			{
				int check2=prob_mutate <= prob_mutation;
				if(check2)
					mutate(parents[0], &mutate_loc[0]);
				clone_town(parents[0], new_pop[j].town, sizeofchromosome);
			}
			else
			{
				crossover(parents, child, grid->cities, cross_loc);

				int check2=prob_mutate <= prob_mutation;
				if(check2)
					mutate(child, &mutate_loc[0]);
				clone_town(child, new_pop[j].town, sizeofchromosome);
			}

		}
		float fit_sum = (float)0.0;
		for (int i=0; i<pop_size; i++)
		{
			new_pop[i].calc_fitness();
			fit_sum        += new_pop[i].fitness;
			new_pop[i].fit_prob = fit_sum;
		}
		// Compute the full probabilities
		for (int i=0; i<pop_size; i++)
			new_pop[i].fit_prob /= fit_sum;

		// Swap the populations
		Grid* temp = old_pop;
		old_pop     = new_pop;
		new_pop     = temp;

		// Select the new leaders
		if (select_leader(old_pop, pop_size, generation_leader, best_leader))
			best_generation = i + 1;
		print_status(generation_leader, best_leader, i + 1);
		count++:
	}
	
	
	free_population(old_pop, population);	
	free_population(new_pop, population);
	free_grid(best_leader); 
	free_grid(generation_leader);

	cout << endl << "No of generations for best generation:  " << best_generation << endl;
}

void selection(Grid* pop, int population, Town** parents, float* rand_nums)
{

	int a=0;
	int b=0;
	while(a<2)
	{
		while(b<2)
		{
			int x=rand_nums[i] <= pop[j].fitness_probability;
			a++;
			b++;

			if(x)
			{
				clone_town(pop[j].town, parents[i], pop[0].cities *  sizeof(Town));
				break;
			}
		}
	}
}

__global__ void fitness_kernel(World* pop, int pop_size)
{
	// Get the thread id
	int tid = getGlobalIdx_2D_1D();
	
	// Evaluate if the thread is valid
	if (tid < pop_size)
	{
		float distance = (float)0.0; // Total "normalized" "distance"
		
		// Calculate fitnesses using the fitness formula that will be explained in the reports
		for (int i=0; i<pop[tid].num_cities-1; i++)
			distance=distance+1;
		pop[tid].fitness = (pop[tid].width * pop[tid].height) / distance;
	}

	int tid = getGlobalIdx_2D_1D();
	
	

	// Get the thread id
	int tid = getGlobalIdx_2D_1D();

	// Evaluate if the thread is valid
	if (tid < pop_size)
		pop[tid].fit_prob /= *fit_sum;

	// Evaluate if the thread is valid
	if (tid < pop_size)
	{
		// Sum of all fitness
		float sum = (float)0.0;
		
		// Calculate the partial sum
		for (int i=0; i<=tid; i++)
			sum += pop[i].fitness;
		pop[tid].fit_prob = sum;

		if (tid == (pop_size - 1))	*fit_sum = sum;
	}


}

void make_grid(Grid* grid, int width, int height, int num_cities, int seed)
{
	set<tuple<int, int>> coordinates;
	set<tuple<int, int>>::iterator it;
	pair<set<tuple<int, int>>::iterator,bool> ret;
	int x;
	int count=0;
	ifstream myfile ("bays29.tsp");
	int arr[58];
  if (myfile.is_open())
  {
    while ( myfile >> x )
    {
    	if(count>=1)
    	{
    		if(count%3==2)
    			arr[(count-2)/3]=x;
    		else if(count%3==0)
    			arr[(count-3)/3]=x;

    	}
      //cout << line << '\n';
      count++;
    }
    myfile.close();
  }
	
	// Create some unique random cities
	for (int i=0; i<num_cities; i++)
	{	
		printf("%d fwf",i);

			printf("hello %f\n",rgen());
			//tuple<int,int> coors((int)(rgen() * width), (int)(rgen() * height));
			tuple<int,int> coors(arr[i], arr[i+1]);
		ret = coordinates.insert(coors);
	}
	
	// Add those cities to the world
	{
		int i = 0;
		for (it=coordinates.begin(); it!=coordinates.end(); it++)
		{
			world->cities[i].x = get<0>(*it);
			world->cities[i].y = get<1>(*it);
			i++;
		}
	}
}

//CPU

void init_world(World* world, int width, int height, int cities)
{
	world->width      = width;
	world->height     = height;
	world->cities = cities;
	world->fitness    = (float)0.0;
	world->fit_prob   = (float)0.0;
	world->city     = new City[cities * sizeof(City)];
}

void clone_city(City* src, City* dst, int cities)
{
memcpy(dst, src, cities * sizeof(City));
}

void clone_world(World* src, World* dst)
{
	dst->width      = src->width;
	dst->height     = src->height;
	dst->cities = src->cities;
	dst->fitness    = src->fitness;
	dst->fit_prob   = src->fit_prob;
	clone_city(src->city, dst->city, src->num_city);
}

void free_world(World* world)
{
	delete[] world->city;
	delete[] world;
}
bool checkForKernelError(const char *err_msg)
{
	hipError_t status = hipGetLastError();
	if (status != hipSuccess)
	{
		cout << err_msg << hipGetErrorString(status) << endl;
		return true;
	}
	else
	{
		return false;
	}
}

void free_population(World* pop, int pop_size)
{
	for (int i=0; i<pop_size; i++)
		delete[] pop[i].city;
	delete[] pop;
}

//GPU

bool g_init_world(World* d_world, World* h_world)
{
	// Error checking
	bool error;
	
	// Soft clone world
	error = g_soft_clone_world(d_world, h_world);
	if (error)
		return true;
	
	// Allocate space for cities on device
	City *d_city;
	error = checkForError(hipMalloc((void**)&d_city, h_world->cities * sizeof(City)));
	if (error)
	return true;
	
	// Update pointer on device
	error = checkForError(hipMemcpy(&d_world->city, &d_city, sizeof(City*), hipMemcpyHostToDevice));
	if (error)
	return true;
	
	return false;
}

bool g_soft_clone_world(World* d_world, World* h_world)
{
	// Error checking
	bool error;
	
	error = checkForError(hipMemcpy(&d_world->width, &h_world->width,        \
		sizeof(int), hipMemcpyHostToDevice));
	if (error)
	return true;
	error = checkForError(hipMemcpy(&d_world->height, &h_world->height,      \
		sizeof(int), hipMemcpyHostToDevice));
	if (error)
	return true;
	error = checkForError(hipMemcpy(&d_world->cities,                    \
		&h_world->cities, sizeof(int), hipMemcpyHostToDevice));
	if (error)
	return true;

	return false;
}

